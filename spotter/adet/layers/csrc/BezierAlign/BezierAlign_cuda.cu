#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

// TODO make it in a common file
#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)


template <typename T>
__device__ T bezier_curve(
    const T p0,
    const T p1,
    const T p2,
    const T p3,
    const T u) {
  return (
      (1. - u) * (1. - u) * (1. - u) * p0
    + 3. * u * (1. - u) * (1. - u) * p1
    + 3. * u * u * (1. - u) * p2
    + u * u * u * p3);
}

template <typename T>
__device__ T bilinear_interpolate(
    const T* bottom_data,
    const int height,
    const int width,
    T y,
    T x,
    const int index /* index for debug only*/) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    // empty
    return 0;
  }

  if (y <= 0)
    y = 0;
  if (x <= 0)
    x = 0;

  int y_low = (int)y;
  int x_low = (int)x;
  int y_high;
  int x_high;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T)y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T)x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;
  // do bilinear interpolation
  T v1 = bottom_data[y_low * width + x_low];
  T v2 = bottom_data[y_low * width + x_high];
  T v3 = bottom_data[y_high * width + x_low];
  T v4 = bottom_data[y_high * width + x_high];
  T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  return val;
}

template <typename T>
__global__ void BezierAlignForward(
    const int nthreads,
    const T* bottom_data,
    const T spatial_scale,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int sampling_ratio,
    const T* bottom_rois,  // bottom rois contains the bezier curve
    T* top_data,
    bool aligned) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    // beziers have size Nx(1+8*2) = Nx17
    const T* offset_bottom_rois = bottom_rois + n * 17;
    int roi_batch_ind = offset_bottom_rois[0];

    // Do not use rounding; this implementation detail is critical
    T offset = aligned ? (T)0.5 : (T)0.0;

    // TODO: avoid this by using parallel annotation, for good
    T p0_x = offset_bottom_rois[1 ] * spatial_scale;
    T p0_y = offset_bottom_rois[2 ] * spatial_scale;
    T p1_x = offset_bottom_rois[3 ] * spatial_scale;
    T p1_y = offset_bottom_rois[4 ] * spatial_scale;
    T p2_x = offset_bottom_rois[5 ] * spatial_scale;
    T p2_y = offset_bottom_rois[6 ] * spatial_scale;
    T p3_x = offset_bottom_rois[7 ] * spatial_scale;
    T p3_y = offset_bottom_rois[8 ] * spatial_scale;
    T p4_x = offset_bottom_rois[15] * spatial_scale;
    T p4_y = offset_bottom_rois[16] * spatial_scale;
    T p5_x = offset_bottom_rois[13] * spatial_scale;
    T p5_y = offset_bottom_rois[14] * spatial_scale;
    T p6_x = offset_bottom_rois[11] * spatial_scale;
    T p6_y = offset_bottom_rois[12] * spatial_scale;
    T p7_x = offset_bottom_rois[9 ] * spatial_scale;
    T p7_y = offset_bottom_rois[10] * spatial_scale;

    // compute the coords
    const T u = pw / static_cast<T>(pooled_width);
    const T v = ph / static_cast<T>(pooled_height);
    const T x0 = bezier_curve(p0_x, p1_x, p2_x, p3_x, u);
    const T y0 = bezier_curve(p0_y, p1_y, p2_y, p3_y, u);
    const T x1 = bezier_curve(p4_x, p5_x, p6_x, p7_x, u);
    const T y1 = bezier_curve(p4_y, p5_y, p6_y, p7_y, u);
    const T x_center = x1 * v + x0 * (1. - v) - offset;
    const T y_center = y1 * v + y0 * (1. - v) - offset;
    
    T roi_width = max(abs(p0_x - p3_x), abs(p4_x - p7_x));
    T roi_height = max(abs(p0_y - p3_y), abs(p4_y - p7_y));
    if (!aligned) { // for backward-compatibility only
      roi_width = max(roi_width, (T)1.);
      roi_height = max(roi_height, (T)1.);
    }
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    const T* offset_bottom_data =
        bottom_data + (roi_batch_ind * channels + c) * height * width;

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0)
        ? sampling_ratio
        : ceil(roi_height / pooled_height); // e.g., = 2
    int roi_bin_grid_w =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    // We do average (integral) pooling inside a bin
    // When the grid is empty, output zeros == 0/1, instead of NaN.
    const T count = max(roi_bin_grid_h * roi_bin_grid_w, 1); // e.g. = 4

    T output_val = 0.;
    for (int iy = 0; iy < roi_bin_grid_h; iy++) // e.g., iy = 0, 1
    {
      const T y = y_center - (T)0.5 * bin_size_h +
          static_cast<T>(iy + .5f) * bin_size_h /
              static_cast<T>(roi_bin_grid_h); // e.g., 0.5, 1.5
      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        const T x = x_center - (T)0.5 * bin_size_w +
            static_cast<T>(ix + .5f) * bin_size_w /
                static_cast<T>(roi_bin_grid_w);

        T val = bilinear_interpolate(
            offset_bottom_data, height, width, y, x, index);
        output_val += val;
      }
    }
    output_val /= count;

    top_data[index] = output_val;
  }
}

template <typename T>
__device__ void bilinear_interpolate_gradient(
    const int height,
    const int width,
    T y,
    T x,
    T& w1,
    T& w2,
    T& w3,
    T& w4,
    int& x_low,
    int& x_high,
    int& y_low,
    int& y_high,
    const int index /* index for debug only*/) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    // empty
    w1 = w2 = w3 = w4 = 0.;
    x_low = x_high = y_low = y_high = -1;
    return;
  }

  if (y <= 0)
    y = 0;
  if (x <= 0)
    x = 0;

  y_low = (int)y;
  x_low = (int)x;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T)y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T)x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;

  // reference in forward
  // T v1 = bottom_data[y_low * width + x_low];
  // T v2 = bottom_data[y_low * width + x_high];
  // T v3 = bottom_data[y_high * width + x_low];
  // T v4 = bottom_data[y_high * width + x_high];
  // T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  return;
}

template <typename T>
__global__ void BezierAlignBackwardFeature(
    const int nthreads,
    const T* top_diff,
    const int num_rois,
    const T spatial_scale,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int sampling_ratio,
    T* bottom_diff,
    const T* bottom_rois,
    bool aligned) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    // beziers have size Nx(1+8*2) = Nx17
    const T* offset_bottom_rois = bottom_rois + n * 17;
    int roi_batch_ind = offset_bottom_rois[0];

    // Do not use rounding; this implementation detail is critical
    T offset = aligned ? (T)0.5 : (T)0.0;
    T p0_x = offset_bottom_rois[1 ] * spatial_scale;
    T p0_y = offset_bottom_rois[2 ] * spatial_scale;
    T p1_x = offset_bottom_rois[3 ] * spatial_scale;
    T p1_y = offset_bottom_rois[4 ] * spatial_scale;
    T p2_x = offset_bottom_rois[5 ] * spatial_scale;
    T p2_y = offset_bottom_rois[6 ] * spatial_scale;
    T p3_x = offset_bottom_rois[7 ] * spatial_scale;
    T p3_y = offset_bottom_rois[8 ] * spatial_scale;
    T p4_x = offset_bottom_rois[15] * spatial_scale;
    T p4_y = offset_bottom_rois[16] * spatial_scale;
    T p5_x = offset_bottom_rois[13] * spatial_scale;
    T p5_y = offset_bottom_rois[14] * spatial_scale;
    T p6_x = offset_bottom_rois[11] * spatial_scale;
    T p6_y = offset_bottom_rois[12] * spatial_scale;
    T p7_x = offset_bottom_rois[9 ] * spatial_scale;
    T p7_y = offset_bottom_rois[10] * spatial_scale;

    // compute the coords
    const T u = pw / static_cast<T>(pooled_width);
    const T v = ph / static_cast<T>(pooled_height);
    const T x0 = bezier_curve(p0_x, p1_x, p2_x, p3_x, u);
    const T y0 = bezier_curve(p0_y, p1_y, p2_y, p3_y, u);
    const T x1 = bezier_curve(p4_x, p5_x, p6_x, p7_x, u);
    const T y1 = bezier_curve(p4_y, p5_y, p6_y, p7_y, u);
    const T x_center = x1 * v + x0 * (1. - v) - offset;
    const T y_center = y1 * v + y0 * (1. - v) - offset;

    T roi_width = max(abs(p0_x - p3_x), abs(p4_x - p7_x));
    T roi_height = max(abs(p0_y - p3_y), abs(p4_y - p7_y));
    if (!aligned) { // for backward-compatibility only
      roi_width = max(roi_width, (T)1.);
      roi_height = max(roi_height, (T)1.);
    }
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    T* offset_bottom_diff =
        bottom_diff + (roi_batch_ind * channels + c) * height * width;

    int top_offset = (n * channels + c) * pooled_height * pooled_width;
    const T* offset_top_diff = top_diff + top_offset;
    const T top_diff_this_bin = offset_top_diff[ph * pooled_width + pw];

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0)
        ? sampling_ratio
        : ceil(roi_height / pooled_height); // e.g., = 2
    int roi_bin_grid_w =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    // We do average (integral) pooling inside a bin
    const T count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

    for (int iy = 0; iy < roi_bin_grid_h; iy++) // e.g., iy = 0, 1
    {
      const T y = y_center - (T)0.5 * bin_size_h +
          static_cast<T>(iy + .5f) * bin_size_h /
              static_cast<T>(roi_bin_grid_h); // e.g., 0.5, 1.5
      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        const T x = x_center - (T)0.5 * bin_size_w +
            static_cast<T>(ix + .5f) * bin_size_w /
                static_cast<T>(roi_bin_grid_w);

        T w1, w2, w3, w4;
        int x_low, x_high, y_low, y_high;

        bilinear_interpolate_gradient(
            height,
            width,
            y,
            x,
            w1,
            w2,
            w3,
            w4,
            x_low,
            x_high,
            y_low,
            y_high,
            index);

        T g1 = top_diff_this_bin * w1 / count;
        T g2 = top_diff_this_bin * w2 / count;
        T g3 = top_diff_this_bin * w3 / count;
        T g4 = top_diff_this_bin * w4 / count;

        if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0) {
          atomicAdd(
              offset_bottom_diff + y_low * width + x_low, static_cast<T>(g1));
          atomicAdd(
              offset_bottom_diff + y_low * width + x_high, static_cast<T>(g2));
          atomicAdd(
              offset_bottom_diff + y_high * width + x_low, static_cast<T>(g3));
          atomicAdd(
              offset_bottom_diff + y_high * width + x_high, static_cast<T>(g4));
        } // if
      } // ix
    } // iy
  } // CUDA_1D_KERNEL_LOOP
} // RoIAlignBackward

namespace adet {

at::Tensor BezierAlign_forward_cuda(
    const at::Tensor& input,
    const at::Tensor& rois,
    const float spatial_scale,
    const int pooled_height,
    const int pooled_width,
    const int sampling_ratio,
    bool aligned) {
  AT_ASSERTM(input.device().is_cuda(), "input must be a CUDA tensor");
  AT_ASSERTM(rois.device().is_cuda(), "rois must be a CUDA tensor");
  at::TensorArg input_t{input, "input", 1}, rois_t{rois, "rois", 2};

  at::CheckedFrom c = "ROIAlign_forward_cuda";
  at::checkAllSameGPU(c, {input_t, rois_t});
  at::checkAllSameType(c, {input_t, rois_t});
  at::cuda::CUDAGuard device_guard(input.device());

  auto num_rois = rois.size(0);
  auto channels = input.size(1);
  auto height = input.size(2);
  auto width = input.size(3);

  auto output = at::empty(
      {num_rois, channels, pooled_height, pooled_width}, input.options());
  auto output_size = num_rois * pooled_height * pooled_width * channels;
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(
      at::cuda::ATenCeilDiv(
          static_cast<int64_t>(output_size), static_cast<int64_t>(512)),
      static_cast<int64_t>(4096)));
  dim3 block(512);

  if (output.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return output;
  }

  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "BezierAlign_forward", [&] {
    BezierAlignForward<scalar_t><<<grid, block, 0, stream>>>(
        output_size,
        input.contiguous().data_ptr<scalar_t>(),
        spatial_scale,
        channels,
        height,
        width,
        pooled_height,
        pooled_width,
        sampling_ratio,
        rois.contiguous().data_ptr<scalar_t>(),
        output.data_ptr<scalar_t>(),
        aligned);
  });
  hipDeviceSynchronize();
  AT_CUDA_CHECK(hipGetLastError());
  return output;
}

// TODO remove the dependency on input and use instead its sizes -> save memory
at::Tensor BezierAlign_backward_cuda(
    const at::Tensor& grad,
    const at::Tensor& rois,
    const float spatial_scale,
    const int pooled_height,
    const int pooled_width,
    const int batch_size,
    const int channels,
    const int height,
    const int width,
    const int sampling_ratio,
    bool aligned) {
  AT_ASSERTM(grad.device().is_cuda(), "grad must be a CUDA tensor");
  AT_ASSERTM(rois.device().is_cuda(), "rois must be a CUDA tensor");

  at::TensorArg grad_t{grad, "grad", 1}, rois_t{rois, "rois", 2};
  at::CheckedFrom c = "ROIAlign_backward_cuda";
  at::checkAllSameGPU(c, {grad_t, rois_t});
  at::checkAllSameType(c, {grad_t, rois_t});
  at::cuda::CUDAGuard device_guard(grad.device());

  auto num_rois = rois.size(0);
  auto grad_input =
      at::zeros({batch_size, channels, height, width}, grad.options());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(
      at::cuda::ATenCeilDiv(
          static_cast<int64_t>(grad.numel()), static_cast<int64_t>(512)),
      static_cast<int64_t>(4096)));
  dim3 block(512);

  // handle possibly empty gradients
  if (grad.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return grad_input;
  }

  AT_DISPATCH_FLOATING_TYPES(grad.scalar_type(), "BezierAlign_backward", [&] {
    BezierAlignBackwardFeature<scalar_t><<<grid, block, 0, stream>>>(
        grad.numel(),
        grad.contiguous().data_ptr<scalar_t>(),
        num_rois,
        spatial_scale,
        channels,
        height,
        width,
        pooled_height,
        pooled_width,
        sampling_ratio,
        grad_input.data_ptr<scalar_t>(),
        rois.contiguous().data_ptr<scalar_t>(),
        aligned);
  });
  AT_CUDA_CHECK(hipGetLastError());
  return grad_input;
}

} // namespace detectron2
